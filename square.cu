
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float *d_out, float *d_in){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    float f = d_in[idx];
    d_out[idx] =  f +1;
}


int main(int argc, char ** argv){
	const int ARRAY_SIZE = 51200;
	const int ARRAY_BYTES =  ARRAY_SIZE * sizeof(float);
    
    float h_in[ARRAY_SIZE],h_out[ARRAY_SIZE];
    for (int i = 0; i<ARRAY_SIZE; i++) {
        h_in[i] =  float(i);
    }
    float * d_in, * d_out;
    // 1. alloc memory on device
    hipMalloc( (void **) &d_in,  ARRAY_BYTES );
    hipMalloc( (void **) &d_out, ARRAY_BYTES );
	

    // 2. htod, launch kernel, dtoh
    hipMemcpy(d_in,  h_in,  ARRAY_BYTES, hipMemcpyHostToDevice);
    square<<< 100,ARRAY_SIZE/100>>> (d_out,d_in);
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    for (int i = 0; i<ARRAY_SIZE; i++) {
        printf("%f ",h_out[i]);
    }
    hipFree(d_in);
    hipFree(d_out);
    
    return 0;
}
